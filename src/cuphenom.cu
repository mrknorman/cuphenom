#include "hip/hip_runtime.h"
#include <inttypes.h>
#include "lal_phenom.h"

int32_t main(){

	const float   mass_1_msun       =   10.0f;
    const float   mass_2_msun       =   10.0f;
    const float   sample_rate_hertz = 8192.0f;
    const int32_t num_samples       = 8192.0f;
    const float   inclination       =    0.0f;
    const float   distance_mpc      =   10.0f;
	
	float2 *strain = NULL;
	
	generateLALInspiral(
		mass_1_msun, 
		mass_2_msun, 
		sample_rate_hertz, 
		num_samples, 
		inclination, 
		distance_mpc, 
		&strain
    );
	
	return 0;
}